#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <cstdlib>
#include <cassert>

#define CHECK_CUDA(call) \
    if ((call) != hipSuccess) { \
        std::cerr << "CUDA error at " << __LINE__ << ": " << hipGetErrorString(hipGetLastError()) << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CHECK_CUSPARSE(call) \
    if ((call) != HIPSPARSE_STATUS_SUCCESS) { \
        std::cerr << "cuSPARSE error at " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

int main() {
    // Load sparse matrix from "matrix.txt" (Matrix Market format)
    std::ifstream fin("matrix.txt");
    if (!fin) {
        std::cerr << "Failed to open matrix.txt" << std::endl;
        return 1;
    }

    std::string line;
    // Skip comments
    do {
        std::getline(fin, line);
    } while (line[0] == '%');

    int numRows, numCols, nnz;
    std::istringstream(line) >> numRows >> numCols >> nnz;

    std::vector<int> h_rowIndices(nnz);
    std::vector<int> h_colIndices(nnz);
    std::vector<float> h_values(nnz);

    for (int i = 0; i < nnz; ++i) {
        int row, col;
        float val;
        fin >> row >> col >> val;
        h_rowIndices[i] = row - 1; // 1-based to 0-based
        h_colIndices[i] = col - 1;
        h_values[i] = val;
    }
    fin.close();

    // Convert COO to CSR
    std::vector<int> h_rowPtr(numRows + 1, 0);
    for (int i = 0; i < nnz; ++i)
        h_rowPtr[h_rowIndices[i] + 1]++;
    for (int i = 0; i < numRows; ++i)
        h_rowPtr[i + 1] += h_rowPtr[i];

    std::vector<int> h_colInd(nnz);
    std::vector<float> h_csrVals(nnz);
    std::vector<int> rowOffset = h_rowPtr;
    for (int i = 0; i < nnz; ++i) {
        int row = h_rowIndices[i];
        int dest = rowOffset[row]++;
        h_colInd[dest] = h_colIndices[i];
        h_csrVals[dest] = h_values[i];
    }

    // Dense matrix dimensions
    int denseCols = 64;
    std::vector<float> h_dense(numCols * denseCols);
    for (auto& val : h_dense)
        val = static_cast<float>(rand()) / RAND_MAX;

    // Allocate device memory
    int *d_rowPtr, *d_colInd;
    float *d_vals, *d_dense, *d_result;
    CHECK_CUDA(hipMalloc((void**)&d_rowPtr, (numRows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_colInd, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals, nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_dense, numCols * denseCols * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_result, numRows * denseCols * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_rowPtr, h_rowPtr.data(), (numRows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_colInd, h_colInd.data(), nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_vals, h_csrVals.data(), nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_dense, h_dense.data(), numCols * denseCols * sizeof(float), hipMemcpyHostToDevice));

    // cuSPARSE setup
    hipsparseHandle_t handle;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, numRows, numCols, nnz,
                                     d_rowPtr, d_colInd, d_vals,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, numCols, denseCols, denseCols,
                                       d_dense, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, numRows, denseCols, denseCols,
                                       d_result, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    float alpha = 1.0f, beta = 0.0f;
    size_t bufferSize = 0;
    void* dBuffer = nullptr;

    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    CHECK_CUSPARSE(hipsparseSpMM(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, dBuffer));

    // Copy result back
    std::vector<float> h_result(numRows * denseCols);
    CHECK_CUDA(hipMemcpy(h_result.data(), d_result, numRows * denseCols * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "Sparse x dense multiplication completed. Example result value: "
              << h_result[0] << std::endl;

    // Cleanup
    hipFree(d_rowPtr);
    hipFree(d_colInd);
    hipFree(d_vals);
    hipFree(d_dense);
    hipFree(d_result);
    hipFree(dBuffer);

    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnMat(matB);
    hipsparseDestroyDnMat(matC);
    hipsparseDestroy(handle);

    return 0;
}
