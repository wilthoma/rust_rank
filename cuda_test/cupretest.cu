/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> // cudaMalloc, cudaMemcpy, etc.
#include <hipsparse.h>         // cusparseSpMM
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include <iostream>
#include <vector>
#include <cassert>
#include <fstream>  // Include fstream for file input

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

void load_sms_matrix(const std::string& filename, std::vector<int>& rowIndices, std::vector<int>& colIndices, std::vector<float>& values, int& numRows, int& numCols, int& nnz) {
    std::ifstream file(filename);  // Make sure to include <fstream>
    if (!file) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(-1);
    }

    char arbitraryChar;
    file >> numRows >> numCols >> arbitraryChar;

    std::vector<int> tempRowIndices;
    std::vector<int> tempColIndices;
    std::vector<float> tempValues;

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        tempRowIndices.push_back(row);
        tempColIndices.push_back(col);
        tempValues.push_back(value);
    }

    nnz = tempRowIndices.size();
    rowIndices = std::move(tempRowIndices);
    colIndices = std::move(tempColIndices);
    values = std::move(tempValues);

    file.close();
}

void coo_matrix_to_csr(int numRows, const std::vector<int>& rowIndices, const std::vector<int>& colIndices, const std::vector<float>& values,
                       std::vector<int>& csrOffsets, std::vector<int>& csrColumns, std::vector<float>& csrValues) {
    csrOffsets.resize(numRows + 1, 0);
    csrColumns.resize(values.size());
    csrValues.resize(values.size());

    std::vector<int> rowCount(numRows, 0);
    for (int i = 0; i < rowIndices.size(); ++i) {
        rowCount[rowIndices[i]]++;
    }

    csrOffsets[0] = 0;
    for (int i = 1; i <= numRows; ++i) {
        csrOffsets[i] = csrOffsets[i - 1] + rowCount[i - 1];
    }

    std::vector<int> tempOffsets = csrOffsets;
    for (int i = 0; i < rowIndices.size(); ++i) {
        int row = rowIndices[i];
        int destIndex = tempOffsets[row]++;
        csrColumns[destIndex] = colIndices[i];
        csrValues[destIndex] = values[i];
    }
}



int main(int argc, char* argv[]) {
    // Host problem definition
    // int   A_num_rows      = 4;
    // int   A_num_cols      = 4;
    // int   A_nnz           = 9;
    // int   B_num_rows      = A_num_cols;
    // int   B_num_cols      = 3;
    // int   ldb             = B_num_rows;
    // int   ldc             = A_num_rows;
    // int   B_size          = ldb * B_num_cols;
    // int   C_size          = ldc * B_num_cols;
    // int   hA_csrOffsets[] = { 0, 3, 4, 7, 9 };
    // int   hA_columns[]    = { 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    // float hA_values[]     = { 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
    //                           6.0f, 7.0f, 8.0f, 9.0f };
    // float hB[]            = { 1.0f,  2.0f,  3.0f,  4.0f,
    //                           5.0f,  6.0f,  7.0f,  8.0f,
    //                           9.0f, 10.0f, 11.0f, 12.0f };
    // float hC[]            = { 0.0f, 0.0f, 0.0f, 0.0f,
    //                           0.0f, 0.0f, 0.0f, 0.0f,
    //                           0.0f, 0.0f, 0.0f, 0.0f };
    // float hC_result[]     = { 19.0f,  8.0f,  51.0f,  52.0f,
    //                           43.0f, 24.0f, 123.0f, 120.0f,
    //                           67.0f, 40.0f, 195.0f, 188.0f };
    // float alpha           = 1.0f;
    // float beta            = 0.0f;

    // load matrix from file
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_file>" << std::endl;
        return -1;
    }
    std::vector<int> rowIndices, colIndices, csrOffsets, csrColumns;
    std::vector<float> values, csrValues;
    int numRows, numCols, nnz;
    load_sms_matrix(argv[1], rowIndices, colIndices, values, numRows, numCols, nnz);
    coo_matrix_to_csr(numRows, rowIndices, colIndices, values, csrOffsets, csrColumns, csrValues);

    std::cout << numRows <<"x" << numCols << " matrix loaded from file: " << argv[1] << " with nnz=" << nnz << std::endl;

    // Random dense matrix for multiplication
    int denseCols = 10;  // Example: Result matrix column size
    std::vector<float> h_dense(numCols * denseCols);
    for (int i = 0; i < numCols * denseCols; ++i) {
        h_dense[i] = 1; //static_cast<float>(rand()) / RAND_MAX;  // Random initialization
    }

    std::vector<float> c_dense(numRows * denseCols);
    for (int i = 0; i < numRows * denseCols; ++i) {
        c_dense[i] = 0; //static_cast<float>(rand()) / RAND_MAX;  // Random initialization
    }

    std::vector<float> c_result(numRows * denseCols);

    int   A_num_rows      = numRows;
    int   A_num_cols      = numCols;
    int   A_nnz           = nnz;
    int   B_num_rows      = A_num_cols;
    int   B_num_cols      = denseCols;
    int   ldb             = B_num_rows;
    int   ldc             = A_num_rows;
    int   B_size          = ldb * B_num_cols;
    int   C_size          = ldc * B_num_cols;
    int* hA_csrOffsets = &csrOffsets[0];
    // std::copy(csrOffsets.begin(), csrOffsets.end(), hA_csrOffsets);
    int*   hA_columns    = &csrColumns[0]; //{ 0, 2, 3, 1, 0, 2, 3, 1, 3 };
    float* hA_values     = &csrValues[0]; //{ 1.0f, 2.0f, 3.0f, 4.0f, 5.0f,
                          //    6.0f, 7.0f, 8.0f, 9.0f };
    float* hB            = &h_dense[0];//{ 1.0f,  2.0f,  3.0f,  4.0f,
                           //   5.0f,  6.0f,  7.0f,  8.0f,
                           //   9.0f, 10.0f, 11.0f, 12.0f };
    float* hC            = &c_dense[0]; //{ 0.0f, 0.0f, 0.0f, 0.0f,
                            //   0.0f, 0.0f, 0.0f, 0.0f,
                            //   0.0f, 0.0f, 0.0f, 0.0f };
    float* hC_result     = &c_result[0]; //{ 19.0f,  8.0f,  51.0f,  52.0f,
                            //   43.0f, 24.0f, 123.0f, 120.0f,
                            //   67.0f, 40.0f, 195.0f, 188.0f };
    float alpha           = 1.0f;
    float beta            = 0.0f;



    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dB, *dC;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))    )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))  )
    CHECK_CUDA( hipMalloc((void**) &dB,         B_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC,         C_size * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));


    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSpMM_preprocess(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "SpMM operation runtime: " << milliseconds << " ms" << std::endl;

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));


    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(float),
                           hipMemcpyDeviceToHost) )
    int correct = 1;
    // for (int i = 0; i < A_num_rows; i++) {
    //     for (int j = 0; j < B_num_cols; j++) {
    //         if (hC[i + j * ldc] != hC_result[i + j * ldc]) {
    //             correct = 0; // direct floating point comparison is not reliable
    //             break;
    //         }
    //     }
    // }
    if (correct)
        printf("spmm_csr_example test PASSED\n");
    else
        printf("spmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC) )
    return EXIT_SUCCESS;
}